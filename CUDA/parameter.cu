
#include "parameter.cuh"

parameter::parameter(char* inputdir):rf()
{
	input_file_name = new char [1024];

	strcpy(input_file_name, inputdir);
    rf.openinput( inputdir ); 

    initial_particle();
	initial_grid();
	initial_static_potential();
	inital_time_set();
};

//////////////////////////////////////////////////////////////////////////////////////////

void parameter::initial_particle( void )
// read and particle information
{

  rf.openinput( input_file_name );

      energy_e      = atof( rf.setget( "&particle", "energy_e" ) );
      energy_ion    = atof( rf.setget( "&particle", "energy_ion"  ) );
      Ib            = atof( rf.setget( "&particle", "Ib" ) );
      wegith        = atof( rf.setget( "&particle", "weigth" ) );
      ni            = atof( rf.setget( "&particle", "atom_density" ) );

  rf.closeinput();

}

//////////////////////////////////////////////////////////////////////////////////////////
void parameter::initial_grid( void )
// read and gridinformation
{

  rf.openinput( input_file_name );

     dr        = atof( rf.setget( "&box", "cells_dr" ) );
     dz        = atof( rf.setget( "&box", "cells_dz"  ) );
	 L         = atof( rf.setget( "&box", "L" ) );
     R         = atof( rf.setget( "&box", "R" ) );	
     ca_rd     = atof( rf.setget( "&box", "cathode_rad" ) );
	 ca_len    = atof( rf.setget( "&box", "cathode_len" ) );
     rf.closeinput();

}
void parameter::initial_static_potential( void )
// read and cell information
{

  rf.openinput( input_file_name );

     anode_p      = atof( rf.setget( "&static_field", "wall_electric_potential" ) );
     cathode_p    = atof( rf.setget( "&static_field", "cathode_electric_potential"  ) );
     screen_p     = atof( rf.setget( "&static_field", "screen_electric_potential" ) );

  rf.closeinput();

}

void parameter::inital_time_set( void )
// read and cell information
{

  rf.openinput( input_file_name );

     total_time      = atof( rf.setget( "&time_set", "total_time" ) );
     time_inter      = atof( rf.setget( "&time_set", "time_interval"  ) );

  rf.closeinput();

}


//EOF

