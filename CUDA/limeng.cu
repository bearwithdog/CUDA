#include "hip/hip_runtime.h"
#include "iostream"
#include "math.h"
#include "fstream"
#include "ostream"
#include "sstream"
#include <iomanip>
#include <vector>
#include <algorithm>
#include <time.h>
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h"
#include"hip/device_functions.h"
#include "common.cuh"
#include"grid.cuh"
#include "particle.cuh"
#include "UPML.cuh"
#include "inteper.cuh"
#include "static_electric.cuh"
#include "static_magnetic.cuh"
#include "lty_paralle_common.cuh"
#include "hiprand/hiprand_kernel.h"
#include <stdlib.h>
#include <stdio.h>
#include "hiprand.h"
#include"variable.cuh"
using namespace std;

int main()
{
	int tmp = 5000;
	hipDeviceReset();
	curandnumber(tmp);
	void data_save(Paticle *pat_elc,Paticle *pat_ion);
	void data_save_ion(Paticle *pat_elc, Paticle *pat_ion);//���ʱ��������Ϣ
	//void data_save(Paticle *pat_elc);
	void data_save(Grid* G_GPU);
	void current_save(Grid* G);
	char* indir="..\\data\\input.txt";
	parameter p(indir);	
	initial_para(p);//��ʼ������
	//cout<<qe<<" "<<Me<<endl;
	 sigmaz1=one_array_malloc(nzz);
	 sigmaz =one_array_malloc(nzz);
	 G_gpu = new Grid[nxx*nzz];//����������֤������ȷ�� ���ݻؿ�
	 pat_elc=new Paticle[lizi_count];//�����ϵ�������Ϣ�����ڽ���GPU����֮��Ľ��
	 pat_ion=new Paticle[lizi_count];//�����ϵ�������Ϣ�����ڽ���GPU����֮��Ľ��
	 hipError_t cudaStatus;
	hipMalloc((void**)&device_G,nxx*nzz*sizeof(Grid));
	hipMalloc((void**)&dev_Gtemp, nxx*nzz*sizeof(Grid));
	hipMalloc((void**)&device_Gn,nxx*nzz*sizeof(Grid));
	//hipMalloc((void**)&device_tail,sizeof(int));//���ں����߳�����
	cudaStatus=hipMalloc((void**)&d_pat_elc,lizi_count*sizeof(Paticle));
	cudaStatus = hipMalloc((void**)&d_pre_elc, lizi_count*sizeof(Pre_Paticle));//�ܵ�ģ��������Ŀ GPU��
	cudaStatus=hipMalloc((void**)&d_pat_ion,lizi_count*sizeof(Paticle));
	cudaStatus = hipMalloc((void**)&d_pre_ion, lizi_count*sizeof(Pre_Paticle));//�ܵ�ģ��������Ŀ GPU��
		if (cudaStatus != hipSuccess) {
	fprintf(stderr, "hipMemcpy failed!");
       }
	cudaStatus=hipMalloc((void**)&d_stac_Bx,nxx*nzz*sizeof(float));
	cudaStatus=hipMalloc((void**)&d_stac_Bz,nxx*nzz*sizeof(float));
	cudaStatus=hipMalloc((void**)&d_stac_ex,nxx*nzz*sizeof(float));
	cudaStatus=hipMalloc((void**)&d_stac_ez,nxx*nzz*sizeof(float));
	read_cross("E:\\cross_section_net.txt");
	 static_magnetic();
	 static_electric();
	 initialsigma();
	cudaStatus=hipMemcpy(d_stac_ex,stac_ex,nxx*nzz*sizeof(float),hipMemcpyHostToDevice);
	cudaStatus=hipMemcpy(d_stac_ez,stac_ez,nxx*nzz*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(d_stac_Bx,stac_Bx,nxx*nzz*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(d_stac_Bz,stac_Bz,nxx*nzz*sizeof(float),hipMemcpyHostToDevice);
	device_initialchang<<<block,thread>>>(device_G,device_Gn);
	kernel_L_InitialPML<<<block,thread>>>(nxx, nzz);
	cudaStatus = hipMalloc((void**)&d_sigmaz1, nzz*sizeof(float));
	cudaStatus = hipMalloc((void**)&d_sigmaz, nzz*sizeof(float));
	cudaStatus = hipMemcpy(d_sigmaz1, sigmaz1, nzz*sizeof(float), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(d_sigmaz, sigmaz, nzz*sizeof(float), hipMemcpyHostToDevice);
	clock_t start=0, end=0;
	
	 ofstream time("D:\\PIC\\time.txt",ios::app);
	 
	initial_always<<<block,thread>>>(d_pat_elc,lizi_count,DS_number,tmp);//���ӳ�ʼ��
	initial_always << <block, thread >> >(d_pat_ion, lizi_count, DS_number,tmp);//���ӳ�ʼ��
	device_define_G << <block,thread >> >(nxx, nzz, device_Gn, device_G);
	start = clock();
	 for(int t=0;t<1;t++)
	 {
		 host_temptail+=30;
		//device_ave_field<<<block,thread>>>(nxx,nzz,device_G,device_Gn);
		device_update_last << <block, thread >> >(d_stac_Bx, d_stac_Bz, d_stac_ex, d_stac_ez, d_pat_elc,d_pre_elc,device_G,DS_number,host_temptail);
		//device_update_ion << <block, thread >> >(d_stac_Bx, d_stac_Bz, d_stac_ex, d_stac_ez, d_pat_ion,d_pre_ion,device_G,DS_number,host_temptail);
		//hipMemcpy(wuchafenxi, (d_pat_elc), 30 * sizeof(Paticle), hipMemcpyDeviceToHost);
		//data_save(wuchafenxi);//ÿ����������30��������Ϣ��Ϊ�ȶԣ�ʵ�ʿ��Բ�Ҫ
		current << <block, thread >> >(d_pat_elc, d_pre_elc, device_G, DS_number, host_temptail,t);
		//current_ion << <block, thread >> >(d_pat_ion, d_pre_ion, device_G, DS_number, host_temptail,t);
		/*hipMemcpy(G_gpu, device_G, nxx*nzz*sizeof(Grid), hipMemcpyDeviceToHost);
		current_save(G_gpu);*/
		cacuchang_hx<<<block,thread>>>(device_G,device_Gn,d_sigmaz1,d_sigmaz,dt,dr,dz,nxx,nzz);
		cacuchang_hy << <block,thread >> >(device_G, device_Gn, d_sigmaz1, d_sigmaz, dt, dr, dz, nxx, nzz);
		cacuchang_hz<< <block, thread >> >(device_G, device_Gn, d_sigmaz1, d_sigmaz, dt, dr, dz, nxx, nzz);
		cacuchang_ex << <block, thread >> >(device_G, device_Gn, d_sigmaz1, d_sigmaz, dt, dr, dz, nxx, nzz);
		cacuchang_ey << <block, thread >> >(device_G, device_Gn, d_sigmaz1, d_sigmaz, dt, dr, dz, nxx, nzz);
		cacuchang_ez << <block, thread >> >(device_G, device_Gn, d_sigmaz1, d_sigmaz, dt, dr, dz, nxx, nzz);
	    device_define_G<<<block,thread>>>(nxx,nzz,device_Gn,device_G);

		/*end = clock();
		cout<<t<<"\t"<<"\t"<<(float)(end - start) / CLOCKS_PER_SEC<<"S"<<endl;
        time<<t<<"  Szabo1 Run time:  "<<(float)(end - start) / CLOCKS_PER_SEC<<"S"<<endl;*/
	 }
	 end = clock();
	 cout<< (float)(end - start) / CLOCKS_PER_SEC << "S" << endl;
	 hipMemcpy(G_gpu,device_G, nxx*nzz*sizeof(Grid), hipMemcpyDeviceToHost);
	 data_save(G_gpu);//���ʱ�̳���Ϣ
	 cudaStatus=hipMemcpy(pat_elc,d_pat_elc,lizi_count*sizeof(Paticle),hipMemcpyDeviceToHost);
	 cudaStatus=hipMemcpy(pat_ion,d_pat_ion,lizi_count*sizeof(Paticle),hipMemcpyDeviceToHost);
	 data_save(pat_elc,pat_ion);//���ʱ��������Ϣ
	 data_save_ion(pat_ion,pat_elc);//���ʱ��������Ϣ
	hipFree(device_G);
	hipFree(device_Gn);
	hipFree(d_pat_elc);
	hipFree(d_pat_ion);
	hipFree(d_rds);
	hipFree(d_rds1);
	hipFree(d_stac_Bx);
	hipFree(d_stac_Bz);
	hipFree(d_stac_ex);
	hipFree(d_stac_ez);
	system("pause");
	return 0;
}

void data_save(Paticle *pat_elc,Paticle *pat_ion)
{
	
	  /*************************************************/
		time_inter=1000;
		//if(t%time_inter==0)//time_inter
		
		char s_ele[1000];  
		sprintf_s(s_ele,"D:\\PIC\\���e\\bingxing1.txt");
		ofstream on_ele(s_ele);	
	    for(int i=0;i<host_temptail;i++)
 		 {		
	        on_ele<<pat_elc[i].pr<<"\t"<<pat_elc[i].py<<"\t"<<pat_elc[i].pz<<"\t"<<pat_elc[i].vr<<"\t"<<pat_elc[i].vy<<"\t"<<pat_elc[i].vz<<endl;
		 }  	
}
void data_save_ion(Paticle *pat_elc, Paticle *pat_ion)
{

	/*************************************************/
	time_inter = 1000;
	//if(t%time_inter==0)//time_inter
	//{
	char s_ele[1000];
	sprintf_s(s_ele, "D:\\PIC\\���e\\bingxing2.txt");
	ofstream on_ele(s_ele);
	for (int i = 0; i<host_temptail; i++)
	{
		on_ele << pat_elc[i].pr << "\t" << pat_elc[i].py << "\t" << pat_elc[i].pz << "\t" << pat_elc[i].vr << "\t" << pat_elc[i].vy << "\t" << pat_elc[i].vz << endl;
	}
}
void data_save(Paticle *pat_elc)
{

	/*************************************************/
	time_inter = 1000;
	char s_ele[1000];
	sprintf_s(s_ele, "D:\\PIC\\���e\\wuchafenxi0.txt");
	ofstream on_ele(s_ele, ofstream::app);
	for (int i = 0; i<30; i++)
	{
		on_ele << pat_elc[i].pr << "\t" << pat_elc[i].py << "\t" << pat_elc[i].pz << "\t" << pat_elc[i].vr << "\t" << pat_elc[i].vy << "\t" << pat_elc[i].vz << endl;
	}

}
void data_save(Grid* G_GPU)
{

	ofstream out_chang("D:\\PIC\\���e\\chang_gpu.txt");
	if (out_chang)
	{
		for (int i = 0; i < nxx*nzz; i++)
			out_chang << G_GPU[i].ex << " " << G_GPU[i].ey << " " << G_GPU[i].ez << " "
			<< G_GPU[i].hx << " " << G_GPU[i].hy << " " << G_GPU[i].hz << endl;
	}
}
void current_save(Grid* G_GPU)
{
	ofstream out_chang("D:\\PIC\\���e\\current_gpu.txt");
	if (out_chang)
	{
		for (int i = 0; i < nxx*nzz; i++)
		out_chang << G_GPU[i].jr << " " << G_GPU[i].jy << " " << G_GPU[i].jz <<endl;
	}
}
//void curandnumber(int n)
//{
//	hipMalloc((void**)&DS_number,n*sizeof(float));
//    HS_number=one_array_malloc(n);
//	srand(1);
//	/*for (int i = 0; i < n; i++)
//	{
//		HS_number[i] = (float)rand() / ((float)RAND_MAX);
//	}*/
//	
//float HS_number[30] = { 0.00125126,0.193304,0.585009,
//		              0.350291,0.82284,0.174108,0.710501,0.303995,0.0914029,
//		              0.147313,0.988525,0.119083,0.0089114,0.531663,0.601764,
//		              0.166234,0.450789,0.0570391,0.783319,0.519883,0.875973,
//					  0.955901,0.539354,0.462081,0.862239,0.779656,0.996796,0.611499,
//					  0.266213,0.840144 };
//hipMemcpy(DS_number,HS_number,n*sizeof(float),hipMemcpyHostToDevice);    
//}
//5000�������
void curandnumber(int n)
{
	hipMalloc((void**)&DS_number, n*sizeof(float));
	HS_number = one_array_malloc(n);
	srand(1);
	for (int i = 0; i < n; i++)
	{
		HS_number[i] = (float)rand() / ((float)RAND_MAX);
	}
	hipMemcpy(DS_number, HS_number, n*sizeof(float), hipMemcpyHostToDevice);
}

