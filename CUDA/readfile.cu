#include <iostream>
#include "readfile.cuh"
using namespace std;
readfile::readfile()
{
  already_open = 0;
  buffer = new char [MAX_LINE_LENGTH];
  if(!buffer){ cout<< "allocation error in readfile"<<endl; exit(0);}
  result = new char [MAX_LINE_LENGTH];
  if(!result){ cout<<"allocation error in readfile"<<endl; exit(0);}
}

void readfile::openinput(char *file)
{
	
	if(!already_open){
		fd=fopen(file,"r+");
      if(fd==NULL){
         cout<<"readfile::openinput: can't open file"<<endl;
         exit(1);
      }
      already_open=1;
   }
   else{
      rewind(fd);
   }
}

void readfile::closeinput( void )
{
   if(already_open){
     int value=fclose( fd );
     already_open=0;
     if(value!=0)
	 {
		 cout<<"file not correctly closed"<<endl;
		 exit(0);
	 }
   }
}

int readfile::setinput(char *a)
{
   int m,n;

   n = (int)strlen(a);

   rewind(fd);

   while(read_one_line()){
      m=(int)strlen(buffer);
      if(m==n){
          if(strncmp(buffer,a,n)==0)return(1);
      }
   }

   return(0);
}

char* readfile::setget(char *key, char *a)
{
   int m,i,n,j=0;

   n = (int)strlen(a);
                                                   // reset file pointer to the key word
   if (!setinput(key)) {
     printf( "\n readfile::setget: key word '%s' missing\n", key );
     exit(-1);
     }

   while(read_one_line()){                        // read lines following the key
     m=(int)strlen(buffer);
     if (strchr(buffer,38)) break;                // '&' contained -> break
     if(m>n+1) {                                  // length sufficient
       for(int i=0;i<m-n;i++) {                       // scan the line for variable name
	 if(strncmp(buffer+i,a,n)==0){            // if found, write it to result[]
	   if(buffer[n+i]=='='){
	     i++;
	     while(buffer[n+i+j]!=',' && n+i+j<m ) {
	       result[j]=buffer[n+i+j];
	       j++;
	     }
	     result[j]=0;
	     return(result);                        // and return pointer to result
	   }
	 }
       }
     }
   }
   printf(" readfile::setget: can't find name ");   // otherwise: send error message
   for(i=0;i<n;i++)putchar(a[i]);
   printf(" following key word %s \n\n",key);
   exit(1);
   return(result);
}

char* readfile::getinput(char *a)
{
   int m,n,i=0,j=0;

   n = (int)strlen(a);

   rewind(fd);

   while(read_one_line()){                        // read lines
     m=(int)strlen(buffer);
     if(m>n+1) {                                  // length sufficient
       for(i=0;i<m-n;i++) {                       // scan the line for variable name
	 if(strncmp(buffer+i,a,n)==0){            // if found, write it to result[]
	   if(buffer[n+i]=='='){
	     i++;
	     while(buffer[n+i+j]!=',' && n+i+j<m ) {
	       result[j]=buffer[n+i+j];
	       j++;
	     }
	     result[j]=0;
	     return(result);                      // and return pointer to result
	   }
	 }
       }
     }
   }
   printf("readfile::getinput: can't find name ");
   for(i=0;i<n;i++)putchar(a[i]);
   printf(" in input file \n");
   exit(1);
   return(result);
}

int readfile::read_one_line( void )
{
   int i=0,c;
   while(i<MAX_LINE_LENGTH){
      c=getc(fd);
      if(c==EOF)return(0);
      else if(c=='\n'){
         buffer[i++]=0;
         return(1);
      }
      else if(c=='#'){
         buffer[i++]=0;
         while(getc(fd)!='\n');
         return(1);
      }
      else if(c!=' '){
         buffer[i++]=c;
      }
   }
   printf("readfile::read_one_line: line too long\n");
   exit(-1);
   return(-1);
}



