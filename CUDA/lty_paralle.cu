#include "hip/hip_runtime.h"
#include "lty_paralle_common.cuh"
//#include "common.cuh"
#include <stdio.h>
#include<iostream>
#include<time.h>
#include"hip/device_functions.h"
#include ""
#include <math.h>
#include "sm_20_atomic_functions.hpp"
using namespace std;
extern int Ne,int nxx,int nzz,int kongxue,int tail,int enlarge;
extern  float dr,float dz,float ca_posz,float KTe;
//extern float *d_ex1,float *d_iex1,float *d_ey1,float *d_iey1,float *d_ez1,float *d_iez1,
//	   float *d_hx1,float *d_ihx1,float *d_hy1,float *d_ihy1,float *d_hz1,float *d_ihz1,
//	   float *d_sigmaz1,float *d_sigmaz,float *sigmaz1,float *sigmaz;
extern Paticle *pat_elc;
__device__ float d_ex1[51 * 101], d_iex1[51 * 101], d_ey1[51 * 101], d_iey1[51 * 101], d_ez1[51 * 101], d_iez1[51 * 101],
d_hx1[51 * 101], d_ihx1[51 * 101], d_hy1[51 * 101], d_ihy1[51 * 101], d_hz1[51 * 101], d_ihz1[51 * 101];
//extern float *d_sigmaz1 = NULL, *d_sigmaz = NULL;
__constant__ float D_parameter[12]={1.0e-003,1.0e-003,5.0e-002,1.0e-001,1.66782e-12,-5.5594e-15,5.5594e-17,0.005,0.03,
									4.095e-16,2.4033e-20,9.1e-31};
//0dr 1dz 2R 3L 4dt 5qe 6qi 7ca_posr 8ca_posz 9KTe 10KTi 11Me /*3.2044e-19*/
__global__ void device_initialchang(Grid* device_g,Grid* device_gn)//x���� y����
{
	int x=51,y=101;
	int tid=blockIdx.x*blockDim.x+threadIdx.x;
	while(tid<x*y)
	{
		(device_g+tid)->ex=0;
		(device_g+tid)->ey=0;
		(device_g+tid)->ez=0;
		(device_g+tid)->hx=0;
		(device_g+tid)->hy=0;
		(device_g+tid)->hz=0;

		(device_g+tid)->ave_ex=0;
		(device_g+tid)->ave_ey=0;
		(device_g+tid)->ave_ez=0;
		(device_g+tid)->ave_hx=0;
		(device_g+tid)->ave_hy=0;
		(device_g+tid)->ave_hz=0;

		(device_g+tid)->ne[0]=0;
		(device_g+tid)->ne[1]=0;
		(device_g+tid)->jr=0.0;
		(device_g+tid)->jz=0.0;
		(device_g+tid)->jy=0.0;
		(device_g+tid)->jr_ion=0.0;
		(device_g+tid)->jz_ion=0.0;
		(device_g+tid)->jy_ion=0.0;
		(device_g+tid)->Pmax=0.0;

		(device_gn+tid)->ex=0;
		(device_gn+tid)->ey=0;
		(device_gn+tid)->ez=0;
		(device_gn+tid)->hx=0;
		(device_gn+tid)->hy=0;
		(device_gn+tid)->hz=0;
		(device_gn+tid)->ne[2]=0;
		tid+=gridDim.x*blockDim.x;
	}
	//printf("success,device_initial_chang");
	
}

//void device_initial_always(float *device_rds,float *device_rds1,Paticle *p_pat_elc,Grid *device_G,float ca_posz,int Ne,int nx,int nz,float dr,float dz,float KTe)
__global__ void initial_always(Paticle *pat_elc,int n,float *S_number,int tmp)//n��step*30
{
	
	int tid=blockIdx.x*blockDim.x+threadIdx.x;
	while(tid<n)
	{
				int i=tid%tmp;
				float afa=0;//rds5*(pi/12);
				float cita=0;//rds6*pi/2;
				float vv = 1e6;
				pat_elc[tid].pz=D_parameter[8];
				pat_elc[tid].pr =0.04*S_number[i]+0.001;//����λ�õ��
				pat_elc[tid].py=2*pi*S_number[i];
				pat_elc[tid].vr = vv*sin(afa)*cos(cita);
				pat_elc[tid].vy = vv*sin(afa)*sin(cita);
				pat_elc[tid].vz=vv;
				//pat_elc[tid].blei = pat_elc[tid].pr/D_parameter[0];
//				pat_elc[tid].blek = pat_elc[tid].pz/D_parameter[1];
				tid+=gridDim.x*blockDim.x;
     }	
	
}

__global__ void device_ave_field(int x,int y,Grid* device_g,Grid* device_gn) //x���� y����
{
	int u=0,v=0;
	int tid=blockIdx.x*blockDim.x+threadIdx.x;
	while(tid<x*(y))
	{

		//bool bv=tid%y;//v;
		//bool bu=tid/(y);//u;
	/*G[u][v].ave_ex=(bv*G[u][v].ex+bu*G[u-1+1-bu][v].ex+(1-bv)*(1-bu)*G[u][v].ex+(1-bv)*bu*G[u][v].ex)/(bv+bu+(1-bv)*(1-bu)+(1-bv)*bu);
	G[u][v].ave_ez=(G[u][v].ez+bv*G[u][v-1+1-bv].ez)/(2-(1-bv));
	G[u][v].ave_hy=(G[u][v].hy+bu*G[u-1+1-bu][v].hy+bv*G[u][v-1+1-bv].hy+bu*bv*G[u-1+1-bu][v-1+1-bv].hy
					+Gn[u][v].hy+bu*Gn[u-1+1-bu][v].hy+bv*Gn[u][v-1+1-bv].hy+bu*bv*Gn[u-1+1-bu][v-1+1-bv].hy)/(8/(3-bu-bv+(1-bu)*(1-bv)));

	G[u][v].ave_ey=G[u][v].ey;
	G[u][v].ave_hx=(G[u][v].hx+Gn[u][v].hx+bv*G[u][v-1+1-bv].hx+bv*Gn[u][v-1+1-bv].hx)/(2+2*bv);
	G[u][v].ave_hz=(G[u][v].hz+Gn[u][v].hz+bu*G[u-1+1-bu][v].hz+bu*Gn[u-1+1-bu][v].hz)/(2+2*bu);*/
	/*device_g[tid].ave_ex=(bv*device_g[tid].ex+bu*device_g[tid-bu*y].ex+(1-bv)*(1-bu)*device_g[tid].ex+(1-bv)*bu*device_g[tid].ex)/(bv+bu+(1-bv)*(1-bu)+(1-bv)*bu);
	device_g[tid].ave_ez=(device_g[tid].ez+bv*device_g[tid-bv].ez)/(2-(1-bv));
	device_g[tid].ave_hy=(device_g[tid].hy+bu*device_g[tid-bu*y].hy+bv*device_g[tid-bv].hy+bu*bv*device_g[tid-bu*y-bv].hy
					+device_gn[tid].hy+bu*device_gn[tid-bu*y].hy+bv*device_gn[tid-bv].hy+bu*bv*device_gn[tid-bu*y-bv].hy)/(8/(3-bu-bv+(1-bu)*(1-bv)));

	device_g[tid].ave_ey=device_g[tid].ey;
	device_g[tid].ave_hx=(device_g[tid].hx+device_gn[tid].hx+bv*device_g[tid-bv].hx+bv*device_gn[tid-bv].hx)/(2+2*bv);
	device_g[tid].ave_hz=(device_g[tid].hz+device_gn[tid].hz+bu*device_g[tid-bu*y].hz+bu*device_gn[tid-bu*y].hz)/(2+2*bu);*/
		u=tid/(y);
		v=tid%y;
		int tid_temp=u*(y)+v;
		if(u==0&&v!=0)
				{
				(*(device_g+tid_temp)).ave_ex=((*(device_g+tid_temp)).ex);
				(*(device_g+tid_temp)).ave_ez=((*(device_g+tid_temp)).ez+(*(device_g+tid_temp-1)).ez)/2;
				(*(device_g+tid_temp)).ave_hy=((*(device_g+tid_temp)).hy+(*(device_g+tid_temp-1)).hy+(*(device_gn+tid_temp)).hy+(*(device_gn+tid_temp-1)).hy)/4;

				(*(device_g+tid_temp)).ave_ey=(*(device_g+tid_temp)).ey;
				(*(device_g+tid_temp)).ave_hx=((*(device_g+tid_temp)).hx+(*(device_gn+tid_temp)).hx+(*(device_g+tid_temp-1)).hx+(*(device_gn+tid_temp-1)).hx)/4;
				(*(device_g+tid_temp)).ave_hz=((*(device_g+tid_temp)).hz+(*(device_gn+tid_temp)).hz)/2;
				}
		else if(v==0&&u!=0)
			 {
				(*(device_g+tid_temp)).ave_ex=((*(device_g+tid_temp)).ex+(*(device_g+tid_temp-y)).ex)/2;
				(*(device_g+tid_temp)).ave_ez=((*(device_g+tid_temp)).ez);
				(*(device_g+tid_temp)).ave_hy=((*(device_g+tid_temp)).hy+(*(device_g+tid_temp-y)).hy+(*(device_gn+tid_temp)).hy+(*(device_gn+tid_temp-y)).hy)/4;
				(*(device_g+tid_temp)).ave_ey=(*(device_g+tid_temp)).ey;
				(*(device_g+tid_temp)).ave_hx=((*(device_g+tid_temp)).hx+(*(device_gn+tid_temp)).hx)/2;
				(*(device_g+tid_temp)).ave_hz=((*(device_g+tid_temp)).hz+(*(device_gn+tid_temp)).hz+(*(device_g+tid_temp-y)).hz+(*(device_gn+tid_temp-y)).hz)/4;
			 }
		else if(v==0&&u==0)
			 {
				(*(device_g+tid_temp)).ave_ex=(*(device_g+tid_temp)).ex;
				(*(device_g+tid_temp)).ave_ez=(*(device_g+tid_temp)).ez;
				(*(device_g+tid_temp)).ave_hy=((*(device_g+tid_temp)).hy+(*(device_gn+tid_temp)).hy)/2;
				(*(device_g+tid_temp)).ave_ey=(*(device_g+tid_temp)).ey;
				(*(device_g+tid_temp)).ave_hx=((*(device_g+tid_temp)).hx+(*(device_gn+tid_temp)).hx)/2;
				(*(device_g+tid_temp)).ave_hz=((*(device_g+tid_temp)).hz+(*(device_gn+tid_temp)).hz)/2;
			 }
		 else{
				(*(device_g+tid_temp)).ave_ex=((*(device_g+tid_temp)).ex+(*(device_g+tid_temp-y)).ex)/2;
				(*(device_g+tid_temp)).ave_ez=((*(device_g+tid_temp)).ez+(*(device_g+tid_temp-1)).ez)/2;
				(*(device_g+tid_temp)).ave_hy=((*(device_g+tid_temp)).hy+(*(device_g+tid_temp-y)).hy+(*(device_g+tid_temp-1)).hy+(*(device_g+tid_temp-y-1)).hy
					+(*(device_gn+tid_temp)).hy+(*(device_gn+tid_temp-y)).hy+(*(device_gn+tid_temp-1)).hy+(*(device_gn+tid_temp-y-1)).hy)/8;
				(*(device_g+tid_temp)).ave_ey=(*(device_g+tid_temp)).ey;
				(*(device_g+tid_temp)).ave_hx=((*(device_g+tid_temp)).hx+(*(device_gn+tid_temp)).hx+(*(device_g+tid_temp-1)).hx+(*(device_gn+tid_temp-1)).hx)/4;
				(*(device_g+tid_temp)).ave_hz=((*(device_g+tid_temp)).hz+(*(device_gn+tid_temp)).hz+(*(device_g+tid_temp-y)).hz+(*(device_gn+tid_temp-y)).hz)/4;
				 }
		   tid+=gridDim.x*blockDim.x;
		  // printf("success,device_initial_chang");
	}		
	
}
__device__ float d_min_1(float x,float y)
{
	return(x<y?x:y);
}
__device__ float d_max_1(float x,float y)
{
	return(x>y?x:y); 
}

__global__ void device_update_last(float *device_static_magneticX, float *device_static_magneticZ, float *device_static_electricX, float *device_static_electricZ, Paticle *p_pat_elc, Pre_Paticle *d_pre_elc, Grid *device_G, float *S_number, int tail)
{
	//printf("%d",tail);
	float  Qm_ion = 7.33945e+5;
	float  Qm = -1.7588e+11;
	float mur = 4.0*pi*1.0e-7;
	float E[3] = { 0, 0, 0 }, B[3] = { 0, 0, 0 };
	int tid = blockDim.x*blockIdx.x + threadIdx.x;
	int threadtid = threadIdx.x;
	//Paticle p_prepat_elc;
	__shared__ float elcVr[thread]; __shared__ float elcVy[thread]; __shared__ float elcVz[thread];
	__shared__ float elcPr[thread]; __shared__ float elcPy[thread]; __shared__ float elcPz[thread];
	__shared__ float PrelcPr[thread];  __shared__ float  PrelcPz[thread];
	while (tid<tail)
	{
		elcVr[threadIdx.x] = p_pat_elc[tid].vr;
		elcVy[threadIdx.x] = p_pat_elc[tid].vy;
		elcVz[threadIdx.x] = p_pat_elc[tid].vz;
		elcPr[threadIdx.x] = p_pat_elc[tid].pr;
		elcPy[threadIdx.x] = p_pat_elc[tid].py;
		elcPz[threadIdx.x] = p_pat_elc[tid].pz;
		
		d_pre_elc[tid].pr = elcPr[threadIdx.x];//
		d_pre_elc[tid].pz = elcPz[threadIdx.x];//
		int ii = (int)(elcPr[threadIdx.x] / D_parameter[0]);               //�Ʒ���
		int kk = (int)(elcPz[threadIdx.x] / D_parameter[1]);
		float wr = (elcPr[threadIdx.x] / D_parameter[0]) - ii;
		float wz = (elcPz[threadIdx.x] / D_parameter[1]) - kk;
		float s1 = (1 - wr)*(1 - wz);
		float s2 = (1 - wr)*wz;
		float s3 = wr*(1 - wz);
		float s4 = wr*wz;
		int grid_temp1 = ii*(nz + 1) + kk;//ii kk
		int grid_temp2 = ii*(nz + 1) + kk + 1;//ii kk+1
		int grid_temp3 = (ii + 1)*(nz + 1) + kk;//ii+1 kk
		int grid_temp4 = (ii + 1)*(nz + 1) + kk + 1;//ii+1 kk+1
		//printf("u:%d\t", tid);
		E[0] = ((*(device_G + grid_temp1)).ave_ex*s1 + (*(device_G + grid_temp3)).ave_ex*s3 + (*(device_G + grid_temp2)).ave_ex*s2 + (*(device_G + grid_temp4)).ave_ex*s4)
			+((*(device_static_electricX+grid_temp1))*s1+(*(device_static_electricX+grid_temp3))*s3+(*(device_static_electricX+grid_temp2))*s2+(*(device_static_electricX+grid_temp4))*s4);//+stac_ex[ii][kk];
		//printf("u:%d\tE0=%f\n", tid,E[0]);
		E[2] = ((*(device_G + grid_temp1)).ave_ez*s1 + (*(device_G + grid_temp3)).ave_ez*s3 + (*(device_G + grid_temp2)).ave_ez*s2 + (*(device_G + grid_temp4)).ave_ez*s4)
			+((*(device_static_electricZ+grid_temp1))*s1+(*(device_static_electricZ+grid_temp3))*s3+(*(device_static_electricZ+grid_temp2))*s2+(*(device_static_electricZ+grid_temp4))*s4);//+(*(device_static_electricZ+grid_temp1));
		//printf("u:%d\tE2=%f\n", tid,E[2]);
		B[1] = (((*(device_G + grid_temp1)).ave_hy*s1 + (*(device_G + grid_temp3)).ave_hy*s3 + (*(device_G + grid_temp2)).ave_hy*s2 + (*(device_G + grid_temp4)).ave_hy*s4)*mur);
		//printf("u:%d\tB1=%f\n", tid, B[1]);
		E[1] = ((*(device_G + grid_temp1)).ave_ey*s1 + (*(device_G + grid_temp3)).ave_ey*s3 + (*(device_G + grid_temp2)).ave_ey*s2 + (*(device_G + grid_temp4)).ave_ey*s4);
		//printf("u:%d\tE1=%f\n", tid,E[1]);
		B[0] = (((*(device_G + grid_temp1)).ave_hx*s1 + (*(device_G + grid_temp3)).ave_hx*s3 + (*(device_G + grid_temp2)).ave_hx*s2 + (*(device_G + grid_temp4)).ave_hx*s4)*mur)
		+(*(device_static_magneticX+grid_temp1))*s1+(*(device_static_magneticX+grid_temp3))*s3+((*(device_static_magneticX+grid_temp2))*s2+(*(device_static_magneticX+grid_temp4))*s4);//+(*(device_static_magneticX+grid_temp3));
		//B[0]=((G[ii][kk].ave_hx*s1+G[ii+1][kk].ave_hx*s3+G[ii][kk+1].ave_hx*s2+G[ii+1][kk+1].ave_hx*s4)*mur)
		//+(stac_Bx[ii][kk] * s1 + stac_Bx[ii + 1][kk] * s3 + stac_Bx[ii][kk + 1] * s2 + stac_Bx[ii + 1][kk + 1] * s4);
		//printf("u:%d\tB0=%f\n", tid, B[0]);
		B[2] = (((*(device_G + grid_temp1)).ave_hz*s1 + (*(device_G + grid_temp3)).ave_hz*s3 + (*(device_G + grid_temp2)).ave_hz*s2 + (*(device_G + grid_temp4)).ave_hz*s4)*mur)
		+((*(device_static_magneticZ+grid_temp1))*s1+(*(device_static_magneticZ+grid_temp3))*s3+(*(device_static_magneticZ+grid_temp2))*s2+(*(device_static_magneticZ+grid_temp4))*s4);//+(*(device_static_magneticZ+grid_temp3));
		//printf("B2=%f\n", B[2]);

		float u1[3] = { 0 }, u2[3] = { 0 }, u3[3] = { 0 };//u_n-1/2,u-,,u+,u_n+1/2    //    electron �ֲ����
		float t[3] = { 0 }, s[3] = { 0 };
		float pp[3][3] = { 0 };

		u1[0] = elcVr[threadIdx.x] + (D_parameter[4] / 2)*Qm*E[0];
		u1[1] = elcVy[threadIdx.x] + (D_parameter[4] / 2)*Qm*E[1];
		u1[2] = elcVz[threadIdx.x] + (D_parameter[4] / 2)*Qm*E[2];

		for (int m = 0; m<3; m++)
		{
			t[m] = (B[m] * Qm*D_parameter[4]) / 2;      // ����˹��ת����t
			s[m] = (2 * t[m]) / (1 + t[m] * t[m]);        //��s
		}
		pp[0][0] = 1 - s[2] * t[2] - s[1] * t[1];      //3*3����Ԫ��
		pp[0][1] = s[1] * t[0] + s[2];
		pp[0][2] = s[2] * t[0] - s[1];
		pp[1][0] = s[0] * t[1] - s[2];
		pp[1][1] = 1 - s[2] * t[2] - s[0] * t[0];
		pp[1][2] = s[0] + s[2] * t[1];
		pp[2][0] = s[0] * t[2] + s[1];
		pp[2][1] = s[1] * t[2] - s[0];
		pp[2][2] = 1 - s[1] * t[1] - s[0] * t[0];

		u2[0] = pp[0][0] * u1[0] + pp[0][1] * u1[1] + pp[0][2] * u1[2];
		u2[1] = pp[1][0] * u1[0] + pp[1][1] * u1[1] + pp[1][2] * u1[2];
		u2[2] = pp[2][0] * u1[0] + pp[2][1] * u1[1] + pp[2][2] * u1[2];

		for (int m = 0; m<3; m++)
			u3[m] = u2[m] + (D_parameter[4] / 2)*Qm*E[m];

		elcVr[threadIdx.x] = u3[0];
		elcVy[threadIdx.x] = u3[1];
		elcVz[threadIdx.x] = u3[2];

		float cit1a = 0;
		if ((elcPr[threadIdx.x]) == 0)
			cit1a = 0;
		else  cit1a = atan((elcVy[threadIdx.x] * D_parameter[4]) / (elcPr[threadIdx.x] + elcVr[threadIdx.x] * D_parameter[4]));
		float temp_x = elcPr[threadIdx.x] + elcVr[threadIdx.x] * D_parameter[4];
		if (temp_x >= 0)
			elcPr[threadIdx.x] = sqrt((elcPr[threadIdx.x] + elcVr[threadIdx.x] * D_parameter[4])*(elcPr[threadIdx.x] + elcVr[threadIdx.x] * D_parameter[4]) + elcVy[threadIdx.x] * D_parameter[4] * elcVy[threadIdx.x] * D_parameter[4]);
		else
		{
			elcPr[threadIdx.x] = -temp_x;
			elcVr[threadIdx.x] = -elcVr[threadIdx.x];
		}
		elcPz[threadIdx.x] = elcPz[threadIdx.x] + elcVz[threadIdx.x] * D_parameter[4];
		elcPy[threadIdx.x] = elcPy[threadIdx.x] + cit1a;
		if ((elcPr[threadIdx.x]) == 0)
		{
			elcVr[threadIdx.x] = elcVr[threadIdx.x];
			elcVy[threadIdx.x] = elcVy[threadIdx.x];
		}
		elcVr[threadIdx.x] = cos(cit1a)*elcVr[threadIdx.x] + sin(cit1a)*elcVy[threadIdx.x];
		elcVy[threadIdx.x] = -sin(cit1a)*elcVr[threadIdx.x] + cos(cit1a)*elcVy[threadIdx.x];
		p_pat_elc[tid].vr = elcVr[threadIdx.x];
		p_pat_elc[tid].vy = elcVy[threadIdx.x];
		p_pat_elc[tid].vz = elcVz[threadIdx.x];
		p_pat_elc[tid].pr = elcPr[threadIdx.x];
		p_pat_elc[tid].py = elcPy[threadIdx.x];
		p_pat_elc[tid].pz = elcPz[threadIdx.x];
		tid += gridDim.x*blockDim.x;
	}
}
__global__ void device_update_ion(float *device_static_magneticX, float *device_static_magneticZ, float *device_static_electricX, float *device_static_electricZ, Paticle *p_pat_elc, Pre_Paticle *d_pre_elc, Grid *device_G, float *S_number, int tail)
{
	//printf("%d",tail);
	float  Qm_ion = 7.33945e+5;
	float  Qm = -1.7588e+11;
	float mur = 4.0*pi*1.0e-7;
	float E[3] = { 0, 0, 0 }, B[3] = { 0, 0, 0 };
	int tid = blockDim.x*blockIdx.x + threadIdx.x;
	int threadtid = threadIdx.x;
	//Paticle p_prepat_elc;
	__shared__ float elcVr[thread]; __shared__ float elcVy[thread]; __shared__ float elcVz[thread];
	__shared__ float elcPr[thread]; __shared__ float elcPy[thread]; __shared__ float elcPz[thread];
	__shared__ float PrelcPr[thread];  __shared__ float  PrelcPz[thread];
	while (tid<tail)
	{
		elcVr[threadIdx.x] = p_pat_elc[tid].vr;
		elcVy[threadIdx.x] = p_pat_elc[tid].vy;
		elcVz[threadIdx.x] = p_pat_elc[tid].vz;
		elcPr[threadIdx.x] = p_pat_elc[tid].pr;
		elcPy[threadIdx.x] = p_pat_elc[tid].py;
		elcPz[threadIdx.x] = p_pat_elc[tid].pz;

		d_pre_elc[tid].pr = elcPr[threadIdx.x];//
		d_pre_elc[tid].pz = elcPz[threadIdx.x];//
		int ii = (int)(elcPr[threadIdx.x] / D_parameter[0]);               //�Ʒ���
		int kk = (int)(elcPz[threadIdx.x] / D_parameter[1]);
		float wr = (elcPr[threadIdx.x] / D_parameter[0]) - ii;
		float wz = (elcPz[threadIdx.x] / D_parameter[1]) - kk;
		float s1 = (1 - wr)*(1 - wz);
		float s2 = (1 - wr)*wz;
		float s3 = wr*(1 - wz);
		float s4 = wr*wz;
		int grid_temp1 = ii*(nz + 1) + kk;//ii kk
		int grid_temp2 = ii*(nz + 1) + kk + 1;//ii kk+1
		int grid_temp3 = (ii + 1)*(nz + 1) + kk;//ii+1 kk
		int grid_temp4 = (ii + 1)*(nz + 1) + kk + 1;//ii+1 kk+1
		//printf("u:%d\t", tid);
		E[0] = ((*(device_G + grid_temp1)).ave_ex*s1 + (*(device_G + grid_temp3)).ave_ex*s3 + (*(device_G + grid_temp2)).ave_ex*s2 + (*(device_G + grid_temp4)).ave_ex*s4)
			+((*(device_static_electricX+grid_temp1))*s1+(*(device_static_electricX+grid_temp3))*s3+(*(device_static_electricX+grid_temp2))*s2+(*(device_static_electricX+grid_temp4))*s4);//+stac_ex[ii][kk];
		//printf("u:%d\tE0=%f\n", tid,E[0]);
		E[2] = ((*(device_G + grid_temp1)).ave_ez*s1 + (*(device_G + grid_temp3)).ave_ez*s3 + (*(device_G + grid_temp2)).ave_ez*s2 + (*(device_G + grid_temp4)).ave_ez*s4)
			+((*(device_static_electricZ+grid_temp1))*s1+(*(device_static_electricZ+grid_temp3))*s3+(*(device_static_electricZ+grid_temp2))*s2+(*(device_static_electricZ+grid_temp4))*s4);//+(*(device_static_electricZ+grid_temp1));
		//printf("u:%d\tE2=%f\n", tid,E[2]);
		B[1] = (((*(device_G + grid_temp1)).ave_hy*s1 + (*(device_G + grid_temp3)).ave_hy*s3 + (*(device_G + grid_temp2)).ave_hy*s2 + (*(device_G + grid_temp4)).ave_hy*s4)*mur);
		//printf("u:%d\tB1=%f\n", tid, B[1]);
		E[1] = ((*(device_G + grid_temp1)).ave_ey*s1 + (*(device_G + grid_temp3)).ave_ey*s3 + (*(device_G + grid_temp2)).ave_ey*s2 + (*(device_G + grid_temp4)).ave_ey*s4);
		//printf("u:%d\tE1=%f\n", tid,E[1]);
		B[0] = (((*(device_G + grid_temp1)).ave_hx*s1 + (*(device_G + grid_temp3)).ave_hx*s3 + (*(device_G + grid_temp2)).ave_hx*s2 + (*(device_G + grid_temp4)).ave_hx*s4)*mur)
		+(*(device_static_magneticX+grid_temp1))*s1+(*(device_static_magneticX+grid_temp3))*s3+((*(device_static_magneticX+grid_temp2))*s2+(*(device_static_magneticX+grid_temp4))*s4);//+(*(device_static_magneticX+grid_temp3));
		//B[0]=((G[ii][kk].ave_hx*s1+G[ii+1][kk].ave_hx*s3+G[ii][kk+1].ave_hx*s2+G[ii+1][kk+1].ave_hx*s4)*mur)
		//+(stac_Bx[ii][kk] * s1 + stac_Bx[ii + 1][kk] * s3 + stac_Bx[ii][kk + 1] * s2 + stac_Bx[ii + 1][kk + 1] * s4);
		//printf("u:%d\tB0=%f\n", tid, B[0]);
		B[2] = (((*(device_G + grid_temp1)).ave_hz*s1 + (*(device_G + grid_temp3)).ave_hz*s3 + (*(device_G + grid_temp2)).ave_hz*s2 + (*(device_G + grid_temp4)).ave_hz*s4)*mur)
		+((*(device_static_magneticZ+grid_temp1))*s1+(*(device_static_magneticZ+grid_temp3))*s3+(*(device_static_magneticZ+grid_temp2))*s2+(*(device_static_magneticZ+grid_temp4))*s4);//+(*(device_static_magneticZ+grid_temp3));
		//printf("B2=%f\n", B[2]);

		float u1[3] = { 0 }, u2[3] = { 0 }, u3[3] = { 0 };//u_n-1/2,u-,,u+,u_n+1/2    //    electron �ֲ����
		float t[3] = { 0 }, s[3] = { 0 };
		float pp[3][3] = { 0 };

		u1[0] = elcVr[threadIdx.x] + (D_parameter[4] / 2)*Qm_ion*E[0];
		u1[1] = elcVy[threadIdx.x] + (D_parameter[4] / 2)*Qm_ion*E[1];
		u1[2] = elcVz[threadIdx.x] + (D_parameter[4] / 2)*Qm_ion*E[2];

		for (int m = 0; m<3; m++)
		{
			t[m] = (B[m] * Qm_ion*D_parameter[4]) / 2;      // ����˹��ת����t
			s[m] = (2 * t[m]) / (1 + t[m] * t[m]);        //��s
		}
		pp[0][0] = 1 - s[2] * t[2] - s[1] * t[1];      //3*3����Ԫ��
		pp[0][1] = s[1] * t[0] + s[2];
		pp[0][2] = s[2] * t[0] - s[1];
		pp[1][0] = s[0] * t[1] - s[2];
		pp[1][1] = 1 - s[2] * t[2] - s[0] * t[0];
		pp[1][2] = s[0] + s[2] * t[1];
		pp[2][0] = s[0] * t[2] + s[1];
		pp[2][1] = s[1] * t[2] - s[0];
		pp[2][2] = 1 - s[1] * t[1] - s[0] * t[0];

		u2[0] = pp[0][0] * u1[0] + pp[0][1] * u1[1] + pp[0][2] * u1[2];
		u2[1] = pp[1][0] * u1[0] + pp[1][1] * u1[1] + pp[1][2] * u1[2];
		u2[2] = pp[2][0] * u1[0] + pp[2][1] * u1[1] + pp[2][2] * u1[2];

		for (int m = 0; m<3; m++)
			u3[m] = u2[m] + (D_parameter[4] / 2)*Qm_ion*E[m];

		elcVr[threadIdx.x] = u3[0];
		elcVy[threadIdx.x] = u3[1];
		elcVz[threadIdx.x] = u3[2];

		float cit1a = 0;
		if ((elcPr[threadIdx.x]) == 0)
			cit1a = 0;
		else  cit1a = atan((elcVy[threadIdx.x] * D_parameter[4]) / (elcPr[threadIdx.x] + elcVr[threadIdx.x] * D_parameter[4]));
		float temp_x = elcPr[threadIdx.x] + elcVr[threadIdx.x] * D_parameter[4];
		if (temp_x >= 0)
			elcPr[threadIdx.x] = sqrt((elcPr[threadIdx.x] + elcVr[threadIdx.x] * D_parameter[4])*(elcPr[threadIdx.x] + elcVr[threadIdx.x] * D_parameter[4]) + elcVy[threadIdx.x] * D_parameter[4] * elcVy[threadIdx.x] * D_parameter[4]);
		else
		{
			elcPr[threadIdx.x] = -temp_x;
			elcVr[threadIdx.x] = -elcVr[threadIdx.x];
		}
		elcPz[threadIdx.x] = elcPz[threadIdx.x] + elcVz[threadIdx.x] * D_parameter[4];
		elcPy[threadIdx.x] = elcPy[threadIdx.x] + cit1a;
		if ((elcPr[threadIdx.x]) == 0)
		{
			elcVr[threadIdx.x] = elcVr[threadIdx.x];
			elcVy[threadIdx.x] = elcVy[threadIdx.x];
		}
		elcVr[threadIdx.x] = cos(cit1a)*elcVr[threadIdx.x] + sin(cit1a)*elcVy[threadIdx.x];
		elcVy[threadIdx.x] = -sin(cit1a)*elcVr[threadIdx.x] + cos(cit1a)*elcVy[threadIdx.x];
		p_pat_elc[tid].vr = elcVr[threadIdx.x];
		p_pat_elc[tid].vy = elcVy[threadIdx.x];
		p_pat_elc[tid].vz = elcVz[threadIdx.x];
		p_pat_elc[tid].pr = elcPr[threadIdx.x];
		p_pat_elc[tid].py = elcPy[threadIdx.x];
		p_pat_elc[tid].pz = elcPz[threadIdx.x];
		tid += gridDim.x*blockDim.x;
	}
}
__global__ void current_ion(Paticle *p_pat_elc, Pre_Paticle *d_pre_elc, Grid *device_G, float *S_number, int tail, int t)
{

	__shared__ float elcVr[thread]; __shared__ float elcVy[thread]; __shared__ float elcVz[thread];
	__shared__ float elcPr[thread]; __shared__ float elcPy[thread]; __shared__ float elcPz[thread];
	__shared__ float PrelcPr[thread]; __shared__ float  PrelcPz[thread];

	int tid = blockDim.x*blockIdx.x + threadIdx.x;
	while (tid < tail)
	{

		elcVy[threadIdx.x] = p_pat_elc[tid].vy;

		elcPr[threadIdx.x] = p_pat_elc[tid].pr;

		elcPz[threadIdx.x] = p_pat_elc[tid].pz;

		PrelcPr[threadIdx.x] = d_pre_elc[tid].pr;//
		PrelcPz[threadIdx.x] = d_pre_elc[tid].pz;//
		if ((elcPz[threadIdx.x] >= D_parameter[3]) || (elcPz[threadIdx.x] < 0) || (elcPr[threadIdx.x] >= D_parameter[2]))
		{
			//int i = tid %30;
			//float afa = 0;//rds5*(pi/12);
			//float cita = 0;//rds6*pi/2;
			//float vv = sqrt(D_parameter[9] * 2 / D_parameter[11]);
			//p_pat_elc[tid].pz = D_parameter[8];
			//p_pat_elc[tid].pr = 0.005*S_number[i];
			//p_pat_elc[tid].py = 2 * pi*S_number[i];
			//p_pat_elc[tid].vr = vv*sin(afa)*cos(cita);
			//p_pat_elc[tid].vy = vv*sin(afa)*sin(cita);
			//p_pat_elc[tid].vz = 3e7;
			//p_pat_elc[tid].blei = p_pat_elc[tid].pr / D_parameter[0];
			//p_pat_elc[tid].blek = p_pat_elc[tid].pz / D_parameter[1];
		}
		else
		{
			int i = (int)(elcPr[threadIdx.x] / D_parameter[0]);
			int j = (int)(elcPz[threadIdx.x] / D_parameter[1]);
			int ii = (int)(PrelcPr[threadIdx.x] / D_parameter[0]);               //�Ʒ���
			int kk = (int)(PrelcPz[threadIdx.x] / D_parameter[1]);
			float wrr = PrelcPr[threadIdx.x] / D_parameter[0] - ii;
			float wzz = PrelcPz[threadIdx.x] / D_parameter[1] - kk;
			float newwrr = elcPr[threadIdx.x] / D_parameter[0] - i;
			float newwzz = elcPz[threadIdx.x] / D_parameter[1] - j;
			float V = abs(pi*((ii + 1)*D_parameter[0] * (ii + 1)*D_parameter[0] - ii*D_parameter[0] * ii*D_parameter[0])*D_parameter[1]);
			float V1 = abs(pi*((i + 1)*D_parameter[0] * (i + 1)*D_parameter[0] - i*D_parameter[0] * i*D_parameter[0])*D_parameter[1]);


			int grid_1 = i*(nz + 1) + j;
			int grid_2 = i*(nz + 1) + j + 1;
			int grid_3 = (i + 1)*(nz + 1) + j;
			int grid_4 = (i + 1)*(nz + 1) + j + 1;
			int grid_temp1 = ii*(nz + 1) + kk;//ii kk
			int grid_temp2 = ii*(nz + 1) + kk + 1;//ii kk+1
			int grid_temp3 = (ii + 1)*(nz + 1) + kk;//ii+1 kk
			int grid_temp4 = (ii + 1)*(nz + 1) + kk + 1;//ii+1 kk+1
			/*(*(device_G + grid_1)).Q += D_parameter[6] * (1 - wrr)*(1 - wzz);
			(*(device_G + grid_2)).Q += D_parameter[6] * (1 - wrr)*wzz;
			(*(device_G + grid_3)).Q += D_parameter[6] * wrr*(1 - wzz);
			(*(device_G + grid_4)).Q += D_parameter[6] * wrr*wzz;*/
			//float area = pi*((ii*D_parameter[0] + D_parameter[0])*(ii*D_parameter[0] + D_parameter[0]) - ii*D_parameter[0] * ii*D_parameter[0])*D_parameter[1];
			/*(*(device_G + grid_1)).den = (*(device_G + grid_1)).Q / area;
			(*(device_G + grid_2)).den = (*(device_G + grid_2)).Q / area;
			(*(device_G + grid_3)).den = (*(device_G + grid_3)).Q / area;
			(*(device_G + grid_4)).den = (*(device_G + grid_4)).Q / area;*/

			//���Ӵ�Խ�����������

			float xp = d_min_1(d_min_1(ii*D_parameter[0], i*D_parameter[0]) + D_parameter[0], d_max_1(d_max_1(ii*D_parameter[0], i*D_parameter[0]), (elcPr[threadIdx.x] + PrelcPr[threadIdx.x]) / 2));
			float zp = d_min_1(d_min_1(kk*D_parameter[1], j*D_parameter[1]) + D_parameter[1], d_max_1(d_max_1(kk*D_parameter[1], j*D_parameter[1]), (elcPz[threadIdx.x] + PrelcPz[threadIdx.x]) / 2));
			float fr1 = D_parameter[6] * (xp - PrelcPr[threadIdx.x]) / D_parameter[4];
			float fz1 = D_parameter[6] * (zp - PrelcPz[threadIdx.x]) / D_parameter[4];

			float fr2 = D_parameter[6] * (elcPr[threadIdx.x] - xp) / D_parameter[4];
			float fz2 = D_parameter[6] * (elcPz[threadIdx.x] - zp) / D_parameter[4];

			float wr1 = (xp + PrelcPr[threadIdx.x]) / 2 / D_parameter[1] - ii;
			float wz1 = (zp + PrelcPz[threadIdx.x]) / 2 / D_parameter[1] - kk;
			float wr2 = (xp + elcPr[threadIdx.x]) / 2 / D_parameter[0] - i;
			float wz2 = (zp + elcPz[threadIdx.x]) / 2 / D_parameter[1] - j;
			//printf("%d,%e,%e\n",tail, wr2, wz2);
			/*float da = (*(device_G + grid_1)).den*p_pat_elc[tid].vz;
			float da1 = (*(device_G + grid_3)).den*p_pat_elc[tid].vz;*/
			///////////////////////////////////////////////////���ӵ����ܶ�	1
			//int logic = ii*i / ((ii - 0.001)*(i - 0.001));//�߼���� ��֧�Ż�
			//float logicV = logic*V1 + (1 - logic)*V;
			if (ii == 0 || i == 0)
			{
				atomicAdd(&((*(device_G + grid_temp1)).jr_ion), (fr1*(1 - wz1) / V));
				__syncthreads();
				atomicAdd(&((*(device_G + grid_temp2)).jr_ion), (fr1*(wz1) / V));
				__syncthreads();
				atomicAdd(&((*(device_G + grid_temp1)).jz_ion), (fz1*(1 - wr1) / V));//ע���Ƿ����i
				__syncthreads();
				atomicAdd(&((*(device_G + grid_temp3)).jz_ion), (fz1*(wr1) / V));
				__syncthreads();

				atomicAdd(&((*(device_G + grid_1)).jr_ion), (fr2*(1 - wz2) / V));
				__syncthreads();
				atomicAdd(&((*(device_G + grid_2)).jr_ion), ((fr2*wz2) / V));
				__syncthreads();
				atomicAdd(&((*(device_G + grid_1)).jz_ion), (fz2*(1 - wr2) / V));
				__syncthreads();
				atomicAdd(&((*(device_G + grid_3)).jz_ion), (fz2*wr2 / V));
				__syncthreads();

			}
			else{
				atomicAdd(&((*(device_G + grid_temp1)).jr_ion), (fr1*(1 - wz1) / V1));
				__syncthreads();
				atomicAdd(&((*(device_G + grid_temp2)).jr_ion), (fr1*(wz1) / V1));
				__syncthreads();
				atomicAdd(&((*(device_G + grid_temp1)).jz_ion), (fz1*(1 - wr1) / V1));//ע���Ƿ����i
				__syncthreads();
				atomicAdd(&((*(device_G + grid_temp3)).jz_ion), (fz1*(wr1) / V1));
				__syncthreads();
				atomicAdd(&((*(device_G + grid_1)).jr_ion), (fr2*(1 - wz2) / V1));
				__syncthreads();
				atomicAdd(&((*(device_G + grid_2)).jr_ion), ((fr2*wz2) / V1));
				__syncthreads();
				atomicAdd(&((*(device_G + grid_1)).jz_ion), (fz2*(1 - wr2) / V1));
				__syncthreads();
				atomicAdd(&((*(device_G + grid_3)).jz_ion), (fz2*wr2 / V1));
				__syncthreads();

			}

			float qedens = D_parameter[6] / V;   //�����ܶ�Jy����
			float Jc = qedens*elcVy[threadIdx.x];
			float mid1, mid2;
			mid1 = xp / D_parameter[0] - i;
			mid2 = zp / D_parameter[1] - j;
			float A1 = (1 - wrr)*(1 - wzz);
			float A3 = wrr*(1 - wzz);
			float A2 = (1 - wrr)*wzz;
			float A4 = wrr*wzz;

			float M1 = (1 - mid1)*(1 - mid2);
			float M3 = mid1*(1 - mid2);
			float M2 = (1 - mid1)*mid2;
			float M4 = mid1*mid2;

			float B1 = (1 - newwrr)*(1 - newwzz);
			float B3 = newwrr*(1 - newwzz);
			float B2 = (1 - newwrr)*newwzz;
			float B4 = newwrr*newwzz;

			atomicAdd(&(*(device_G + grid_temp1)).jy_ion, (Jc*(A1 + M1)) / 4 - 0.0);
			__syncthreads();
			atomicAdd(&(*(device_G + grid_temp2)).jy_ion, (Jc*(A2 + M2)) / 4 - 0.0);
			__syncthreads();
			atomicAdd(&(*(device_G + grid_temp3)).jy_ion, (Jc*(A3 + M3)) / 4 - 0.0);
			__syncthreads();
			atomicAdd(&(*(device_G + grid_temp4)).jy_ion, (Jc*(A4 + M4)) / 4 - 0.0);
			__syncthreads();

			atomicAdd(&(*(device_G + grid_1)).jy_ion, (Jc*(B1 + M1)) / 4 - 0.0);
			__syncthreads();
			atomicAdd(&(*(device_G + grid_2)).jy_ion, (Jc*(B2 + M2)) / 4 - 0.0);
			__syncthreads();
			atomicAdd(&(*(device_G + grid_3)).jy_ion, (Jc*(B3 + M3)) / 4 - 0.0);
			__syncthreads();
			atomicAdd(&(*(device_G + grid_4)).jy_ion, (Jc*(B4 + M4)) / 4 - 0.0);
			__syncthreads();
		}
		tid += gridDim.x*blockDim.x;
	}
}
__global__ void current(Paticle *p_pat_elc, Pre_Paticle *d_pre_elc, Grid *device_G, float *S_number, int tail,int t)
{

	__shared__ float elcVr[thread]; __shared__ float elcVy[thread]; __shared__ float elcVz[thread];
	__shared__ float elcPr[thread]; __shared__ float elcPy[thread]; __shared__ float elcPz[thread];
	__shared__ float PrelcPr[thread];__shared__ float  PrelcPz[thread];

	int tid = blockDim.x*blockIdx.x + threadIdx.x;
	while (tid < tail)
	{
		
		elcVy[threadIdx.x] = p_pat_elc[tid].vy;
		
		elcPr[threadIdx.x] = p_pat_elc[tid].pr;

		elcPz[threadIdx.x] = p_pat_elc[tid].pz;

		PrelcPr[threadIdx.x] = d_pre_elc[tid].pr;//
		PrelcPz[threadIdx.x] = d_pre_elc[tid].pz;//
		if ((elcPz[threadIdx.x] >= D_parameter[3]) || (elcPz[threadIdx.x] < 0) || (elcPr[threadIdx.x] >= D_parameter[2]))
		{
			//int i = tid %30;
			//float afa = 0;//rds5*(pi/12);
			//float cita = 0;//rds6*pi/2;
			//float vv = sqrt(D_parameter[9] * 2 / D_parameter[11]);
			//p_pat_elc[tid].pz = D_parameter[8];
			//p_pat_elc[tid].pr = 0.005*S_number[i];
			//p_pat_elc[tid].py = 2 * pi*S_number[i];
			//p_pat_elc[tid].vr = vv*sin(afa)*cos(cita);
			//p_pat_elc[tid].vy = vv*sin(afa)*sin(cita);
			//p_pat_elc[tid].vz = 3e7;
			//p_pat_elc[tid].blei = p_pat_elc[tid].pr / D_parameter[0];
			//p_pat_elc[tid].blek = p_pat_elc[tid].pz / D_parameter[1];
		}
		else
		{
			int i = (int)(elcPr[threadIdx.x] / D_parameter[0]);
			int j = (int)(elcPz[threadIdx.x] / D_parameter[1]);
			int ii = (int)(PrelcPr[threadIdx.x] / D_parameter[0]);               //�Ʒ���
			int kk = (int)(PrelcPz[threadIdx.x] / D_parameter[1]);
			float wrr = PrelcPr[threadIdx.x] / D_parameter[0] - ii;
			float wzz = PrelcPz[threadIdx.x] / D_parameter[1] - kk;
			float newwrr = elcPr[threadIdx.x] / D_parameter[0] - i;
			float newwzz = elcPz[threadIdx.x] / D_parameter[1] - j;
			float V = abs(pi*((ii + 1)*D_parameter[0] * (ii + 1)*D_parameter[0] - ii*D_parameter[0] * ii*D_parameter[0])*D_parameter[1]);
			float V1 = abs(pi*((i + 1)*D_parameter[0] * (i + 1)*D_parameter[0] - i*D_parameter[0] * i*D_parameter[0])*D_parameter[1]);
			
			
			int grid_1 = i*(nz + 1) + j;
			int grid_2 = i*(nz + 1) + j + 1;
			int grid_3 = (i + 1)*(nz + 1) + j;
			int grid_4 = (i + 1)*(nz + 1) + j + 1;
			int grid_temp1 = ii*(nz + 1) + kk;//ii kk
			int grid_temp2 = ii*(nz + 1) + kk + 1;//ii kk+1
			int grid_temp3 = (ii + 1)*(nz + 1) + kk;//ii+1 kk
			int grid_temp4 = (ii + 1)*(nz + 1) + kk + 1;//ii+1 kk+1
			/*(*(device_G + grid_1)).Q += D_parameter[5] * (1 - wrr)*(1 - wzz);
			(*(device_G + grid_2)).Q += D_parameter[5] * (1 - wrr)*wzz;
			(*(device_G + grid_3)).Q += D_parameter[5] * wrr*(1 - wzz);
			(*(device_G + grid_4)).Q += D_parameter[5] * wrr*wzz;*/
			//float area = pi*((ii*D_parameter[0] + D_parameter[0])*(ii*D_parameter[0] + D_parameter[0]) - ii*D_parameter[0] * ii*D_parameter[0])*D_parameter[1];
			/*(*(device_G + grid_1)).den = (*(device_G + grid_1)).Q / area;
			(*(device_G + grid_2)).den = (*(device_G + grid_2)).Q / area;
			(*(device_G + grid_3)).den = (*(device_G + grid_3)).Q / area;
			(*(device_G + grid_4)).den = (*(device_G + grid_4)).Q / area;*/

			//���Ӵ�Խ�����������

			float xp = d_min_1(d_min_1(ii*D_parameter[0], i*D_parameter[0]) + D_parameter[0], d_max_1(d_max_1(ii*D_parameter[0], i*D_parameter[0]), (elcPr[threadIdx.x] + PrelcPr[threadIdx.x]) / 2));
			float zp = d_min_1(d_min_1(kk*D_parameter[1], j*D_parameter[1]) + D_parameter[1], d_max_1(d_max_1(kk*D_parameter[1], j*D_parameter[1]), (elcPz[threadIdx.x] + PrelcPz[threadIdx.x]) / 2));
			float fr1 = D_parameter[5] * (xp - PrelcPr[threadIdx.x]) / D_parameter[4];
			float fz1 = D_parameter[5] * (zp - PrelcPz[threadIdx.x]) / D_parameter[4];
			
			float fr2 = D_parameter[5] * (elcPr[threadIdx.x] - xp) / D_parameter[4];
			float fz2 = D_parameter[5] * (elcPz[threadIdx.x] - zp) / D_parameter[4];
			
			float wr1 = (xp + PrelcPr[threadIdx.x]) / 2 / D_parameter[1] - ii;
			float wz1 = (zp + PrelcPz[threadIdx.x]) / 2 / D_parameter[1] - kk;
			float wr2 = (xp + elcPr[threadIdx.x]) / 2 / D_parameter[0] - i;
			float wz2 = (zp + elcPz[threadIdx.x]) / 2 / D_parameter[1] - j;
			//printf("%d,%e,%e\n",tail, wr2, wz2);
			/*float da = (*(device_G + grid_1)).den*p_pat_elc[tid].vz;
			float da1 = (*(device_G + grid_3)).den*p_pat_elc[tid].vz;*/
			///////////////////////////////////////////////////���ӵ����ܶ�	1
			//int logic = ii*i / ((ii - 0.001)*(i - 0.001));//�߼���� ��֧�Ż�
			//float logicV = logic*V1 + (1 - logic)*V;
			if (ii == 0||i==0 )
			{
				atomicAdd(&((*(device_G + grid_temp1)).jr), (fr1*(1 - wz1) / V));
				__syncthreads();
				atomicAdd(&((*(device_G + grid_temp2)).jr), (fr1*(wz1) / V));
				__syncthreads();
				atomicAdd(&((*(device_G + grid_temp1)).jz), (fz1*(1 - wr1) / V));//ע���Ƿ����i
				__syncthreads();
				atomicAdd(&((*(device_G + grid_temp3)).jz), (fz1*(wr1) / V));
				__syncthreads();
				
				atomicAdd(&((*(device_G + grid_1)).jr), (fr2*(1 - wz2) / V));
				__syncthreads();
				atomicAdd(&((*(device_G + grid_2)).jr), ((fr2*wz2) / V));
				__syncthreads();
				atomicAdd(&((*(device_G + grid_1)).jz), (fz2*(1 - wr2) / V));
				__syncthreads();
				atomicAdd(&((*(device_G + grid_3)).jz), (fz2*wr2 / V));
				__syncthreads();

			}
			else{
				atomicAdd(&((*(device_G + grid_temp1)).jr), (fr1*(1 - wz1) / V1));
				__syncthreads();
				atomicAdd(&((*(device_G + grid_temp2)).jr), (fr1*(wz1) / V1));
				__syncthreads();
				atomicAdd(&((*(device_G + grid_temp1)).jz), (fz1*(1 - wr1) / V1));//ע���Ƿ����i
				__syncthreads();
				atomicAdd(&((*(device_G + grid_temp3)).jz), (fz1*(wr1) / V1));
				__syncthreads();
				atomicAdd(&((*(device_G + grid_1)).jr), (fr2*(1 - wz2) / V1));
				__syncthreads();
				atomicAdd(&((*(device_G + grid_2)).jr), ((fr2*wz2) / V1));
				__syncthreads();
				atomicAdd(&((*(device_G + grid_1)).jz), (fz2*(1 - wr2) / V1));
				__syncthreads();
				atomicAdd(&((*(device_G + grid_3)).jz), (fz2*wr2 / V1));
				__syncthreads();
				
			}
	
			float qedens = D_parameter[5] / V;   //�����ܶ�Jy����
			float Jc = qedens*elcVy[threadIdx.x];
			float mid1, mid2;
			mid1 = xp / D_parameter[0] - i;
			mid2 = zp / D_parameter[1] - j;
			float A1 = (1 - wrr)*(1 - wzz);
			float A3 = wrr*(1 - wzz);
			float A2 = (1 - wrr)*wzz;
			float A4 = wrr*wzz;

			float M1 = (1 - mid1)*(1 - mid2);
			float M3 = mid1*(1 - mid2);
			float M2 = (1 - mid1)*mid2;
			float M4 = mid1*mid2;

			float B1 = (1 - newwrr)*(1 - newwzz);
			float B3 = newwrr*(1 - newwzz);
			float B2 = (1 - newwrr)*newwzz;
			float B4 = newwrr*newwzz;
			
			atomicAdd(&(*(device_G + grid_temp1)).jy, (Jc*(A1 + M1)) / 4-0.0);
			__syncthreads();
			atomicAdd(&(*(device_G + grid_temp2)).jy, (Jc*(A2 + M2)) / 4-0.0);
			__syncthreads();
			atomicAdd(&(*(device_G + grid_temp3)).jy, (Jc*(A3 + M3)) / 4-0.0);
			__syncthreads();
			atomicAdd(&(*(device_G + grid_temp4)).jy, (Jc*(A4 + M4)) / 4-0.0);
			__syncthreads();
		
			atomicAdd(&(*(device_G + grid_1)).jy, (Jc*(B1 + M1)) / 4-0.0);
			__syncthreads();
			atomicAdd(&(*(device_G + grid_2)).jy, (Jc*(B2 + M2)) / 4-0.0);
			__syncthreads();
			atomicAdd(&(*(device_G + grid_3)).jy, (Jc*(B3 + M3)) / 4-0.0);
			__syncthreads();
			atomicAdd(&(*(device_G + grid_4)).jy, (Jc*(B4 + M4)) / 4-0.0);
			__syncthreads();
		}
		tid += gridDim.x*blockDim.x;
	}
}

__global__ void device_define_G(int n, int m, Grid *device_gn, Grid *device_g)
{
	
		int tid=blockIdx.x*blockDim.x+threadIdx.x;
		while(tid<n*m)
		{
			/*int i=tid/m;
			int j=tid%m;
			int tid_temp=i*(m+1)+j;*/
			(device_gn+tid)->ey=(device_g+tid)->ey;
			(device_gn+tid)->hx=(device_g+tid)->hx;
			(device_gn+tid)->hz=(device_g+tid)->hz;
			(device_gn+tid)->ex=(device_g+tid)->ex;
			(device_gn+tid)->hy=(device_g+tid)->hy;
			(device_gn+tid)->ez=(device_g+tid)->ez;
			(device_g+tid)->ne[2]=0;

			(device_g+tid)->jr=0.0;
			(device_g+tid)->jz=0.0;
			(device_g+tid)->jy=0.0;
			(device_g+tid)->Q=0.0;
			(device_g+tid)->den=0.0;

			(device_g+tid)->jr_ion=0.0;
			(device_g+tid)->jz_ion=0.0;
			(device_g+tid)->jy_ion=0.0;
			(device_g+tid)->Q_ion=0.0;
			(device_g+tid)->den_ion=0.0;
			tid+=gridDim.x*blockDim.x;
		}
		
		/* printf("success,device_define_G");*/
}

__device__ void L_InitialPML(int tid)
{
	
	    d_ex1[tid] = 0;
	    d_iex1[tid] = 0;
		d_ey1[tid] = 0;
		d_iey1[tid] = 0;
		d_ez1[tid] = 0;
		d_iez1[tid] = 0;
		d_hx1[tid] = 0;
		d_ihx1[tid] = 0; 
		d_hy1[tid]=0; 
		d_ihy1[tid]=0; 
		d_hz1[tid]=0; 
		d_ihz1[tid]=0;
		
}

__global__ void kernel_L_InitialPML(int nxx, int nzz)
{
	int tid = blockDim.x*blockIdx.x + threadIdx.x;
	while (tid<nxx*nzz)
	{
		L_InitialPML(tid);
		tid += gridDim.x*blockDim.x;
	}
	
}
__global__ void cacuchang_hx(Grid *device_G,Grid *device_Gn,float *d_sigmaz1,float *d_sigmaz,float dt,float dr,float dz,int nxx,int nzz)              
{
	
	float epsl=(8.854e-12);
	float mur=4.0*pi*1.0e-7;
	//float t0=120*dt,T=100*dt;
	float /*ca=0.0,*//*cb=0.0,*/ca1=0.0,cb1=0.0/*,ca2=0,cb2=0*/;
	float /*da=0.0,db=0.0,*/da1=0.0,db1=0.0,da2=0,db2=0;
	int tid=blockDim.x*blockIdx.x+threadIdx.x;
	/////********************   TE01    ******************/////
	
	while (tid<nxx*nzz)
	{
		
				int i=tid/nzz;
				int k=tid%nzz;
				if (i != nx&&k != nz)
				{
					da2 = 1;//(1+dt*sigmaz1[k]/epsl/2);
					db2 = 1;//(1-dt*sigmaz1[k]/epsl/2);
					da1 = (2 * epsl - dt*d_sigmaz[k]) / (2 * epsl + dt*d_sigmaz[k]);
					db1 = (2 * epsl) / (2 * epsl + d_sigmaz[k] * dt);
					d_hx1[tid] = da1*d_hx1[tid] + db1*dt*((*(device_Gn + tid+1)).ey - (*(device_Gn + tid)).ey) / dz;
					(*(device_G + tid)).hx = (*(device_Gn + tid)).hx + (d_hx1[tid] - d_ihx1[tid]) / mur;
					d_ihx1[tid] = d_hx1[tid];
				}
			tid+=gridDim.x*blockDim.x;
	}
	
}
__global__ void cacuchang_hy(Grid *device_G, Grid *device_Gn, float *d_sigmaz1, float *d_sigmaz, float dt, float dr, float dz, int nxx, int nzz)
{

	float epsl = (8.854e-12);
	float mur = 4.0*pi*1.0e-7;
	//float t0=120*dt,T=100*dt;
	float /*ca=0.0,*//*cb=0.0,*/ca1 = 0.0, cb1 = 0.0/*,ca2=0,cb2=0*/;
	float /*da=0.0,db=0.0,*/da1 = 0.0, db1 = 0.0, da2 = 0, db2 = 0;
	int tid = blockDim.x*blockIdx.x + threadIdx.x;
	/////********************   TE01    ******************/////

	while (tid<nxx*nzz)
	{

		int i = tid / nzz;
		int k = tid%nzz;
		if (i != nx&&k != nz)
		{
			da2 = 1;//(1+dt*sigmaz1[k]/epsl/2);
			db2 = 1;//(1-dt*sigmaz1[k]/epsl/2);
			ca1 = (2 * epsl - dt*d_sigmaz[k]) / (2 * epsl + dt*d_sigmaz[k]);
			cb1 = 2 * epsl / (2 * epsl + dt*d_sigmaz[k]);
			d_hy1[tid] = d_hy1[tid] + dt*(((*(device_Gn + tid+nzz)).ez - (*(device_Gn + tid)).ez) / dr - ((*(device_Gn + tid+1)).ex - (*(device_Gn + tid)).ex) / dz);
			(*(device_G + tid)).hy = ca1*(*(device_Gn + tid)).hy + cb1*(d_hy1[tid] - d_ihy1[tid]) / mur;
			d_ihy1[tid] = d_hy1[tid];
		}
		tid += gridDim.x*blockDim.x;
	}

}
__global__ void cacuchang_hz(Grid *device_G, Grid *device_Gn, float *d_sigmaz1, float *d_sigmaz, float dt, float dr, float dz, int nxx, int nzz)
{

	float epsl = (8.854e-12);
	float mur = 4.0*pi*1.0e-7;
	//float t0=120*dt,T=100*dt;
	float /*ca=0.0,*//*cb=0.0,*/ca1 = 0.0, cb1 = 0.0/*,ca2=0,cb2=0*/;
	float /*da=0.0,db=0.0,*/da1 = 0.0, db1 = 0.0, da2 = 0, db2 = 0;
	int tid = blockDim.x*blockIdx.x + threadIdx.x;
	/////********************   TE01    ******************/////
	while (tid<nxx*nzz)
	{

		int i = tid / nzz;
		int k = tid%nzz;
		if (i != nx&&k != nz)
		{
			da2 = 1;//(1+dt*sigmaz1[k]/epsl/2);
			db2 = 1;//(1-dt*sigmaz1[k]/epsl/2);
			d_hz1[tid] = d_hz1[tid] - dt*((*(device_Gn + tid+nzz)).ey - (*(device_Gn + tid)).ey) / dr - dt*((*(device_Gn + tid+nzz)).ey + (*(device_Gn + tid)).ey) / (2 * (i + 0.5)*dr);
			(*(device_G + tid)).hz = (*(device_Gn + tid)).hz + (da2*d_hz1[tid] - db2*d_ihz1[tid]) / mur;
			d_ihz1[tid] = d_hz1[tid];
		}
		tid += gridDim.x*blockDim.x;
	}

}

__global__ void cacuchang_ex(Grid *device_G,Grid *device_Gn,float *d_sigmaz1,float *d_sigmaz,float dt,float dr,float dz,int nxx,int nzz)   
{
	
	float epsl=(8.854e-12);
	const float cgm=0.0;
	float mur=4.0*pi*1.0e-7;
	float A,B,C,D,v,a6=0.0;
	//float bate=0;
	float omega=0,freq=0;
	float t0=120*dt,T=100*dt;
	float ca=0.0,cb=0.0,ca1=0.0,cb1=0.0,ca2=0,cb2=0;
	float da=0.0,db=0.0,da1=0.0,db1=0.0,da2=0,db2=0;

    v=1/sqrt(mur*epsl);
	A=(2*epsl-cgm*dt)/(2*epsl-cgm*dt);
	B=(2*dt)/(2*epsl+cgm*dt);
	C=1;
	D=dt/mur;	
	freq=1e8;
	int tid = blockDim.x*blockIdx.x + threadIdx.x;
	while (tid < nxx*nzz)
	{
		int i = tid / nzz;
		int k = tid%nzz;
		if (i != nx && k != nz)
		{
			if (k == 0)
			{
				float v = 1 / sqrt(mur*epsl);
				float coff = (v*dt - dz) / (v*dt + dz);		
				device_G[tid].ex = device_Gn[tid + 1].ex + coff*(device_Gn[tid + 1].ex - device_Gn[tid].ex);
			}
			else
			{
				ca = (2 * epsl - dt*d_sigmaz1[k]) / (2 * epsl + dt*d_sigmaz1[k]);
				cb = 2 * dt / (2 * epsl + dt*d_sigmaz1[k]);
				d_ex1[tid] = ca*d_ex1[tid] - cb*(device_G[tid].hy - device_G[tid - 1].hy) / dz;
				device_G[tid].ex = device_Gn[tid].ex + (d_ex1[tid] - d_iex1[tid]) - B*(device_G[tid].jr + device_G[tid].jr_ion);
				d_iex1[tid] = d_ex1[tid];
			}
		}
		tid += gridDim.x*blockDim.x;
	}
		
	
}

__global__ void cacuchang_ey(Grid *device_G,Grid *device_Gn,float *d_sigmaz1,float *d_sigmaz,float dt,float dr,float dz,int nxx,int nzz)   
{
	
	float epsl=(8.854e-12);
	const float cgm=0.0;
	float mur=4.0*pi*1.0e-7;
	float A,B,C,D,v,a6=0.0;
	float omega=0,freq=0;
	float t0=120*dt,T=100*dt;
	float ca=0.0,cb=0.0,ca1=0.0,cb1=0.0,ca2=0,cb2=0;
	float da=0.0,db=0.0,da1=0.0,db1=0.0,da2=0,db2=0;

    v=1/sqrt(mur*epsl);
	A=(2*epsl-cgm*dt)/(2*epsl-cgm*dt);
	B=(2*dt)/(2*epsl+cgm*dt);
	C=1;
	D=dt/mur;	
	freq=1e8;
	int tid = blockDim.x*blockIdx.x + threadIdx.x;
	while(tid<nxx*nzz)
	{
		int i=tid/nz;
		int k=tid%nz;
		if (i != nx&&k != nz)
		{
			int tid1 = i*nzz + k;
			int tid2 = i*nzz + k + 1;
			int tid3 = (i + 1)*nzz + k;
			int tid4 = (i + 1)*nzz + k + 1;
			if (i == 0)
			{
				device_G[tid].ey = 0;//te01
			}
			else  if (k == 0)
			{
				float v = 1 / sqrt(mur*epsl);
				float coff = (v*dt - dz) / (v*dt + dz);
				device_G[tid].ey = device_Gn[tid + 1].ey + coff*(device_Gn[tid + 1].ey - device_Gn[tid].ey);//TE01

			}
			else
			{
				da = (2 * epsl - dt*d_sigmaz1[k]) / (2 * epsl + dt*d_sigmaz1[k]); //TE01
				db = 2 / (2 * epsl + dt*d_sigmaz1[k]);
				d_ey1[tid] = d_ey1[tid] + dt*((device_G[tid].hx - device_G[tid - 1].hx) / dz - (device_G[tid].hz - device_G[tid-nzz].hz) / dr);//�˴����ش���
				device_G[tid].ey = da*device_Gn[tid].ey + db*(d_ey1[tid] - d_iey1[tid]) - B*(device_G[tid].jy + device_G[tid].jy_ion) * 10;
				d_iey1[tid] = d_ey1[tid];
			}
		}
			 tid+=gridDim.x*blockDim.x;
	}
	
}
__global__ void cacuchang_ez(Grid *device_G,Grid *device_Gn,float *d_sigmaz1,float *d_sigmaz,float dt,float dr,float dz,int nxx,int nzz)
{
	
	float epsl=(8.854e-12);
	const float cgm=0.0;
	float mur=4.0*pi*1.0e-7;
	float A,B,C,D,v,a6=0.0;
	//float bate=0;
	float omega=0,freq=0;
	float t0=120*dt,T=100*dt;
	float ca=0.0,cb=0.0,ca1=0.0,cb1=0.0,ca2=0,cb2=0;
	float da=0.0,db=0.0,da1=0.0,db1=0.0,da2=0,db2=0;

    v=1/sqrt(mur*epsl);
	A=(2*epsl-cgm*dt)/(2*epsl-cgm*dt);
	B=(2*dt)/(2*epsl+cgm*dt);
	//printf("%e\n",B);
	C=1;
	D=dt/mur;	
	freq=1e8;
	int tid = blockDim.x*blockIdx.x + threadIdx.x;
	while(tid<nxx*nzz)
	{
		int i=tid/nzz;
		int k=tid%nzz;
		if (i != nx&&k != nz)
		{
			if (i == 0)
			{
				ca2 = 1;//(2*epsl+dt*sigmaz[k])/(2*epsl);
				cb2 = 1;//(2*epsl-dt*sigmaz[k])/(2*epsl);
				d_ez1[tid] = d_ez1[tid] + dt * 4 * (*(device_G + tid)).hy / dr / epsl;
				(*(device_G + tid)).ez = (*(device_Gn + tid)).ez + (ca2*d_ez1[tid] - cb2*d_iez1[tid]) - B*((*(device_G + tid)).jz + (*(device_G + tid)).jz_ion);
				d_iez1[tid] = d_ez1[tid];
			}
			else
			{
				ca2 = 1;//(2*epsl+dt*sigmaz[k])/(2*epsl);//TM01
				cb2 = 1;//(2*epsl-dt*sigmaz[k])/(2*epsl);
				d_ez1[tid] = d_ez1[tid] + dt*((1 / (2 * i*dr) + 1 / dr)*(*(device_G + tid)).hy + ((1 / (2 * i*dr)) - 1 / dr)*(*(device_G + tid - nzz)).hy);
				(*(device_G + tid)).ez = (*(device_Gn + tid)).ez + (ca2*d_ez1[tid] - cb2*d_iez1[tid]) / epsl - B*((*(device_G + tid)).jz + (*(device_G + tid)).jz_ion);
				d_iez1[tid] = d_ez1[tid];
			}
		}
	 tid+=gridDim.x*blockDim.x;
	}
}
